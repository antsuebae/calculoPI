#include <cstdio>
#include <cstdlib>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <hip/hip_runtime.h>

__global__ void montecarlo_kernel(long long total_points, unsigned long long *d_count, unsigned int seed) {
    unsigned long long local_count = 0;
    long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    long long stride = gridDim.x * blockDim.x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    for (long long i = idx; i < total_points; i += stride) {
        float x = hiprand_uniform(&state);
        float y = hiprand_uniform(&state);
        if (x * x + y * y <= 1.0f)
            local_count++;
    }

    atomicAdd(d_count, local_count);
}

int main(int argc, char **argv) {
    if (argc != 2) {
        std::fprintf(stderr, "Uso: %s <num_puntos>\n", argv[0]);
        return EXIT_FAILURE;
    }

    long long total_points = std::atoll(argv[1]);
    unsigned long long *d_count, h_count = 0;

    hipMalloc(&d_count, sizeof(unsigned long long));
    hipMemset(d_count, 0, sizeof(unsigned long long));

    int threads_per_block = 256;
    int blocks = 128;  // Puedes ajustar esto según la GPU

    auto start = std::chrono::high_resolution_clock::now();
    montecarlo_kernel<<<blocks, threads_per_block>>>(total_points, d_count, time(NULL));
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    hipMemcpy(&h_count, d_count, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    hipFree(d_count);

    double pi = 4.0 * static_cast<double>(h_count) / static_cast<double>(total_points);
    double tiempo = std::chrono::duration<double>(end - start).count();

    // Salida estructurada
    std::printf("\nResultados - CUDA Monte Carlo\n");
    std::printf("---------------------------------\n");
    std::printf("Puntos totales: %lld\n", total_points);
    std::printf("Puntos dentro del círculo: %llu\n", h_count);
    std::printf("Tiempo de ejecución: %.6f segundos\n", tiempo);
    std::printf("Estimación de π: %.12f\n", pi);
    std::printf("---------------------------------\n");

    return 0;
}