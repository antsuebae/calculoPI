#include <cstdio>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>

__global__ void pi_seq_kernel(long long N, double *result) {
    double h = 1.0 / static_cast<double>(N);
    double sum = 0.0;
    for (long long i = 0; i < N; ++i) {
        double x = (i + 0.5) * h;
        sum += 4.0 / (1.0 + x * x);
    }
    *result = sum * h;
}

int main(int argc, char **argv) {
    if (argc != 2) {
        std::fprintf(stderr, "Uso: %s <num_muestras>\n", argv[0]);
        return EXIT_FAILURE;
    }

    long long muestras = std::atoll(argv[1]);
    double *d_result = nullptr;
    hipMalloc(&d_result, sizeof(double));

    auto start = std::chrono::high_resolution_clock::now();
    pi_seq_kernel<<<1, 1>>>(muestras, d_result);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    double pi = 0.0;
    hipMemcpy(&pi, d_result, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_result);

    double tiempo = std::chrono::duration<double>(end - start).count();

    // Salida con formato estructurado
    std::printf("\nResultados - CUDA Secuencial\n");
    std::printf("---------------------------------\n");
    std::printf("Muestras: %llu\n", muestras);
    std::printf("Tiempo de ejecución: %.6f segundos\n", tiempo);
    std::printf("Estimación de π: %.12f\n", pi);
    std::printf("---------------------------------\n");

    return 0;
}